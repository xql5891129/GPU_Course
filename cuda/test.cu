#include "hip/hip_runtime.h"
// 这是程序二的模板程序，我们已经准备好了加载数据集和加载程序一模型参数的部分，请实现CUDA的深度学习推理过程，请严格保持输出格式输出
// 编译的命令为：nvcc test.cu -o test -Xcompiler "-O3 -std=c++14" -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70
// 编译的命令为：nvcc test.cu -o test -g -G -Xcompiler "-O3 -std=c++14" -gencode arch=compute_50,code=sm_50 -gencode arch=compute_52,code=sm_52 -gencode arch=compute_53,code=sm_53 -gencode arch=compute_60,code=sm_60 -gencode arch=compute_61,code=sm_61 -gencode arch=compute_62,code=sm_62 -gencode arch=compute_70,code=sm_70 -lhdf5 -lhdf5_cpp 

#include <iostream>
#include <vector>
#include <chrono>
#include <iomanip>
#include <string>
#include <fstream>
#include <map>
#include <dirent.h>
#include <cstring>
#include <hdf5/serial/H5Cpp.h>

#define wbCheck(stmt)  do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "\n\nFailed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s \n\n", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

/****************************************************************************************
 * 读取模型参数
 ****************************************************************************************/
// 获取目录中的所有 .txt 文件
std::vector<std::string> get_files_in_directory(const std::string& dir) {
    std::vector<std::string> files;
    DIR* dp;
    struct dirent* entry;
    if ((dp = opendir(dir.c_str())) != NULL) {
        while ((entry = readdir(dp)) != NULL) {
            std::string filename = entry->d_name;
            if (filename.find(".txt") != std::string::npos) {
                files.push_back(filename);
            }
        }
        closedir(dp);
    } else {
        perror("opendir");
    }
    return files;
}

// 读取 .txt 文件并转换为 std::vector<float>
std::vector<float> read_param(const std::string& filepath) {
    std::vector<float> data;
    std::ifstream file(filepath);
    if (file.is_open()) {
        float value;
        while (file >> value) {
            data.push_back(value);
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filepath << std::endl;
    }
    return data;
}

std::map<std::string, std::vector<float>> read_params(std::string dir) {
    // std::string dir = "."; // 当前目录
    std::map<std::string, std::vector<float>> params;

    // 获取目录中的所有 .txt 文件
    std::vector<std::string> param_files = get_files_in_directory(dir);
    for (const auto& file : param_files) {
        std::string filename = file.substr(0, file.find_last_of(".")); // 获取不带扩展名的文件名
        params[filename] = read_param(dir + "/" + file);
    }

    // // 访问参数时可以使用 params["conv1_weight"]
    // for (const auto& kv : params) {
    //     std::cout << "Key: " << kv.first << ", Values: ";
    //     // for (const auto& value : kv.second) {
    //     //     std::cout << value << " ";
    //     // }
    //     std::cout << std::endl;
    // }

    return params;
}

/****************************************************************************************
 * 读取训练集数据
 ****************************************************************************************/

using namespace H5;
void read_h5_file(const std::string& file_path, std::vector<std::vector<float>>& list_of_points, std::vector<int>& list_of_labels) {
    try {
        // 打开文件
        H5File file(file_path, H5F_ACC_RDONLY);

        // 获取文件中的所有数据集名称
        std::vector<std::string> dataset_names;
        hsize_t num_objs = file.getNumObjs();
        for (hsize_t i = 0; i < num_objs; i++) {
            dataset_names.push_back(file.getObjnameByIdx(i));
        }

        // 读取每个数据集
        for (const auto& name : dataset_names) {
            DataSet dataset = file.openDataSet(name + "/points");
            DataSpace dataspace = dataset.getSpace();

            // 获取数据集的维度
            hsize_t dims[2];
            dataspace.getSimpleExtentDims(dims, NULL);

            // 读取数据
            std::vector<float> points(dims[0] * dims[1]);
            dataset.read(points.data(), PredType::NATIVE_FLOAT);

            // 存储点云数据
            list_of_points.push_back(points);

            // 读取标签
            Attribute label_attr = file.openGroup(name).openAttribute("label");
            int label;
            label_attr.read(PredType::NATIVE_INT, &label);

            // 存储标签
            list_of_labels.push_back(label);
        }
    } catch (FileIException& error) {
        error.printErrorStack();
    } catch (DataSetIException& error) {
        error.printErrorStack();
    } catch (DataSpaceIException& error) {
        error.printErrorStack();
    } catch (DataTypeIException& error) {
        error.printErrorStack();
    }
}




// 范例kernel函数，无实际作用
__global__ void add_arrays(int* a, int* b, int* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}



// 用于打印测试
void printVector(float* a,int length)
{

    for(int j=0;j<length;j++){
        std::cout <<a[j]<<std::endl;
    }

}


//矩阵转置
__global__ void matrixTranspose(float* input, float* output, int input_row, int input_col)
{
      int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在矩阵范围内
    if (point_idx < input_row * input_col)
     {
        int row = point_idx / input_col; // 当前输入矩阵的行
        int col = point_idx % input_col; // 当前输入矩阵的列
        output[col * input_row + row] = input[point_idx]; // 转置操作
    }
}


//矩阵乘法
__global__ void matrixMultiply(float* A, float* B, float* output, int M, int N, int K) {
    // A: MxN matrix
    // B: NxK matrix
    // output: MxK result matrix
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(point_idx < M * K) 
    {
        // 计算当前行、列
        int row = point_idx / K;
        int col = point_idx % K;

        float value = 0.0f;
        for(int i = 0; i < N; ++i) {
            value += A[row * N + i] * B[i * K + col];
        }
        output[point_idx] = value;
    }
}


//Convolution卷积操作
__global__ void Convolution(float* input, float* output, float* weight, float* bias, int num_points, int in_channels, int out_channels) 
{
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(point_idx < num_points * out_channels)
    {
        // 计算当前行、列
        int row = point_idx / num_points;
        int col = point_idx % num_points;
        output[point_idx] = bias[row];
        for (int i = 0; i < in_channels; i++) 
        {
            output[point_idx] += input[i * num_points + col] * weight[row * in_channels + i];
       }
         
    }
    
}


//BatchNorm批归一化操作
__global__ void BatchNorm(float* input, float* output, float* weight, float* bias, float* mean, float* var, int num_points, int feat_stn_bn1_var_channels)
{
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //计算当前point_idx所在行
    int row = point_idx / num_points;
    if(point_idx < num_points * feat_stn_bn1_var_channels)
    {
         // 归一化
        output[point_idx] = weight[row] * (input[point_idx] - mean[row]) / sqrtf(var[row] + 1e-5f) + bias[row];
    }  

}

//Relu激活函数操作
__global__ void Relu(float* input, float* output,  int num_points, int feat_stn_relu_channels)
 {
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (point_idx < num_points * feat_stn_relu_channels)
    {
        if (input[point_idx] <= 0)
            output[point_idx] = 0;
        else
            output[point_idx] = input[point_idx];
    }
}

//MaxPool最大池化操作
__global__ void MaxPool(float* input, float* output,int num_points,int feat_stn_max_channels)
{
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;


    if (point_idx < feat_stn_max_channels)
    {
        float max = input[point_idx * num_points];
        for (int i = 0; i < num_points;i+=100)
        {
            if(input[point_idx * num_points + i] > max)
            {
                max = input[point_idx * num_points + i];
            }
        }
        output[point_idx]=max;
    }
}


//feat_stn_fc
__global__ void FC(float* input, float* output,float* weight, float* bias,int in_channels, int out_channels) 
{
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(point_idx < out_channels)
    {
        //初始化为偏置
        output[point_idx] = bias[point_idx];

        for (int i = 0; i < in_channels; i++)
        {
            output[point_idx] += input[i] * weight[point_idx * in_channels + i];
        } 
    }
}

//Iden仿射变换
__global__ void Iden(float* input, int out_h,int out_w)
{
    int point_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(point_idx < out_h * out_w)
    {
        // 计算当前行、列
        int row = point_idx / out_w;
        int col = point_idx % out_w;
        if(row == col)
        {
            input[point_idx] += 1;
        }
    }
}



int main(int argc, char *argv[]) {
    
    std::string dir = argv[1];  // 第一个参数是程序所在的目录，这个目录是存放前一步训练模型参数文件的目录，从这个目录下读取模型参数文件，相对于这个目录读取测试集点云数据和标签
    // cout << dir;
    
    // 读取模型参数
    std::map<std::string, std::vector<float>> params = read_params(dir);

    std::vector<float> feat_stn_conv1_weight_host = params["feat.stn.conv1.weight"];
    std::vector<float> feat_stn_conv1_bias_host = params["feat.stn.conv1.bias"];
    std::vector<float> feat_stn_bn1_weight_host = params["feat.stn.bn1.weight"];
    std::vector<float> feat_stn_bn1_bias_host = params["feat.stn.bn1.bias"];
    std::vector<float> feat_stn_bn1_mean_host = params["feat.stn.bn1.running_mean"];
    std::vector<float> feat_stn_bn1_var_host = params["feat.stn.bn1.running_var"];

    std::vector<float> feat_stn_conv2_weight_host = params["feat.stn.conv2.weight"];
    std::vector<float> feat_stn_conv2_bias_host = params["feat.stn.conv2.bias"];
    std::vector<float> feat_stn_bn2_weight_host = params["feat.stn.bn2.weight"];
    std::vector<float> feat_stn_bn2_bias_host = params["feat.stn.bn2.bias"];
    std::vector<float> feat_stn_bn2_mean_host = params["feat.stn.bn2.running_mean"];
    std::vector<float> feat_stn_bn2_var_host = params["feat.stn.bn2.running_var"];

    std::vector<float> feat_stn_conv3_weight_host = params["feat.stn.conv3.weight"];
    std::vector<float> feat_stn_conv3_bias_host = params["feat.stn.conv3.bias"];
    std::vector<float> feat_stn_bn3_weight_host = params["feat.stn.bn3.weight"];
    std::vector<float> feat_stn_bn3_bias_host = params["feat.stn.bn3.bias"];
    std::vector<float> feat_stn_bn3_mean_host = params["feat.stn.bn3.running_mean"];
    std::vector<float> feat_stn_bn3_var_host = params["feat.stn.bn3.running_var"];

    std::vector<float> feat_stn_fc1_weight_host = params["feat.stn.fc1.weight"];
    std::vector<float> feat_stn_fc1_bias_host = params["feat.stn.fc1.bias"];
    std::vector<float> feat_stn_bn4_weight_host = params["feat.stn.bn4.weight"];
    std::vector<float> feat_stn_bn4_bias_host = params["feat.stn.bn4.bias"];
    std::vector<float> feat_stn_bn4_mean_host = params["feat.stn.bn4.running_mean"];
    std::vector<float> feat_stn_bn4_var_host = params["feat.stn.bn4.running_var"];

    std::vector<float> feat_stn_fc2_weight_host = params["feat.stn.fc2.weight"];
    std::vector<float> feat_stn_fc2_bias_host = params["feat.stn.fc2.bias"];
    std::vector<float> feat_stn_bn5_weight_host = params["feat.stn.bn5.weight"];
    std::vector<float> feat_stn_bn5_bias_host = params["feat.stn.bn5.bias"];
    std::vector<float> feat_stn_bn5_mean_host = params["feat.stn.bn5.running_mean"];
    std::vector<float> feat_stn_bn5_var_host = params["feat.stn.bn5.running_var"];

    std::vector<float> feat_stn_fc3_weight_host = params["feat.stn.fc3.weight"];
    std::vector<float> feat_stn_fc3_bias_host = params["feat.stn.fc3.bias"];

    std::vector<float> feat_conv1_weight_host = params["feat.conv1.weight"];
    std::vector<float> feat_conv1_bias_host = params["feat.conv1.bias"];
    std::vector<float> feat_bn1_weight_host = params["feat.bn1.weight"];
    std::vector<float> feat_bn1_bias_host = params["feat.bn1.bias"];
    std::vector<float> feat_bn1_mean_host = params["feat.bn1.running_mean"];
    std::vector<float> feat_bn1_var_host = params["feat.bn1.running_var"];




    std::vector<float> feat_fstn_conv1_weight_host = params["feat.fstn.conv1.weight"];
    std::vector<float> feat_fstn_conv1_bias_host = params["feat.fstn.conv1.bias"];
    std::vector<float> feat_fstn_bn1_weight_host = params["feat.fstn.bn1.weight"];
    std::vector<float> feat_fstn_bn1_bias_host = params["feat.fstn.bn1.bias"];
    std::vector<float> feat_fstn_bn1_mean_host = params["feat.fstn.bn1.running_mean"];
    std::vector<float> feat_fstn_bn1_var_host = params["feat.fstn.bn1.running_var"];

    std::vector<float> feat_fstn_conv2_weight_host = params["feat.fstn.conv2.weight"];
    std::vector<float> feat_fstn_conv2_bias_host = params["feat.fstn.conv2.bias"];
    std::vector<float> feat_fstn_bn2_weight_host = params["feat.fstn.bn2.weight"];
    std::vector<float> feat_fstn_bn2_bias_host = params["feat.fstn.bn2.bias"];
    std::vector<float> feat_fstn_bn2_mean_host = params["feat.fstn.bn2.running_mean"];
    std::vector<float> feat_fstn_bn2_var_host = params["feat.fstn.bn2.running_var"];

    std::vector<float> feat_fstn_conv3_weight_host = params["feat.fstn.conv3.weight"];
    std::vector<float> feat_fstn_conv3_bias_host = params["feat.fstn.conv3.bias"];
    std::vector<float> feat_fstn_bn3_weight_host = params["feat.fstn.bn3.weight"];
    std::vector<float> feat_fstn_bn3_bias_host = params["feat.fstn.bn3.bias"];
    std::vector<float> feat_fstn_bn3_mean_host = params["feat.fstn.bn3.running_mean"];
    std::vector<float> feat_fstn_bn3_var_host = params["feat.fstn.bn3.running_var"];

    std::vector<float> feat_fstn_fc1_weight_host = params["feat.fstn.fc1.weight"];
    std::vector<float> feat_fstn_fc1_bias_host = params["feat.fstn.fc1.bias"];
    std::vector<float> feat_fstn_bn4_weight_host = params["feat.fstn.bn4.weight"];
    std::vector<float> feat_fstn_bn4_bias_host = params["feat.fstn.bn4.bias"];
    std::vector<float> feat_fstn_bn4_mean_host = params["feat.fstn.bn4.running_mean"];
    std::vector<float> feat_fstn_bn4_var_host = params["feat.fstn.bn4.running_var"];

    std::vector<float> feat_fstn_fc2_weight_host = params["feat.fstn.fc2.weight"];
    std::vector<float> feat_fstn_fc2_bias_host = params["feat.fstn.fc2.bias"];
    std::vector<float> feat_fstn_bn5_weight_host = params["feat.fstn.bn5.weight"];
    std::vector<float> feat_fstn_bn5_bias_host = params["feat.fstn.bn5.bias"];
    std::vector<float> feat_fstn_bn5_mean_host = params["feat.fstn.bn5.running_mean"];
    std::vector<float> feat_fstn_bn5_var_host = params["feat.fstn.bn5.running_var"];

    std::vector<float> feat_fstn_fc3_weight_host = params["feat.fstn.fc3.weight"];
    std::vector<float> feat_fstn_fc3_bias_host = params["feat.fstn.fc3.bias"];
    
    std::vector<float> feat_conv2_weight_host = params["feat.conv2.weight"];
    std::vector<float> feat_conv2_bias_host = params["feat.conv2.bias"];
    std::vector<float> feat_bn2_weight_host = params["feat.bn2.weight"];
    std::vector<float> feat_bn2_bias_host = params["feat.bn2.bias"];
    std::vector<float> feat_bn2_mean_host = params["feat.bn2.running_mean"];
    std::vector<float> feat_bn2_var_host = params["feat.bn2.running_var"];

    std::vector<float> feat_conv3_weight_host = params["feat.conv3.weight"];
    std::vector<float> feat_conv3_bias_host = params["feat.conv3.bias"];
    std::vector<float> feat_bn3_weight_host = params["feat.bn3.weight"];
    std::vector<float> feat_bn3_bias_host = params["feat.bn3.bias"];
    std::vector<float> feat_bn3_mean_host = params["feat.bn3.running_mean"];
    std::vector<float> feat_bn3_var_host = params["feat.bn3.running_var"];

    std::vector<float> fc1_weight_host_host = params["fc1.weight"];
    std::vector<float> fc1_bias_host_host = params["fc1.bias"];
    std::vector<float> bn1_weight_host = params["bn1.weight"];
    std::vector<float> bn1_bias_host = params["bn1.bias"];
    std::vector<float> bn1_mean_host = params["bn1.running_mean"];
    std::vector<float> bn1_var_host = params["bn1.running_var"];

    std::vector<float> fc2_weight_host = params["fc2.weight"];
    std::vector<float> fc2_bias_host = params["fc2.bias"];
    std::vector<float> bn2_weight_host = params["bn2.weight"];
    std::vector<float> bn2_bias_host = params["bn2.bias"];
    std::vector<float> bn2_mean_host = params["bn2.running_mean"];
    std::vector<float> bn2_var_host = params["bn2.running_var"];

    std::vector<float> fc3_weight_host = params["fc3.weight"];
    std::vector<float> fc3_bias_host = params["fc3.bias"];




    //打印测试
    // for(const auto& value :params["bn1.weight"]) {
    //     std::cout<<value;
    // 

    std::string file_path = "./data/test_point_clouds.h5";
    std::vector<std::vector<float>> list_of_points;
    std::vector<int> list_of_labels;
    // 读取训练集数据
    read_h5_file(file_path, list_of_points, list_of_labels);

    
    int result = 0;//记录最后输出的结果
    int num = 0;   //记录正确识别数量

    int max_points = 34800;

    float * input;
	hipMalloc((void**)& input, sizeof(float) * max_points*3);
	float * Tinput;
	hipMalloc((void**)& Tinput, sizeof(float) * max_points*3);
    float * output64;
    hipMalloc((void**)& output64, sizeof(float) * max_points*64);
    float * output64_2;
    hipMalloc((void**)& output64_2, sizeof(float) * max_points*64);
    float * output64_3;
    hipMalloc((void**)& output64_3, sizeof(float) * max_points*64);   
    float * Toutput64;
    hipMalloc((void**)& Toutput64, sizeof(float) * max_points*64);    
    float * Toutput64_2;
    hipMalloc((void**)& Toutput64_2, sizeof(float) * max_points*64);
    float * Toutput64_3;
    hipMalloc((void**)& Toutput64_3, sizeof(float) * max_points*64);  
    float * output128;
    hipMalloc((void**)& output128, sizeof(float) * max_points*128);
    float * output128_1;
    hipMalloc((void**)& output128_1, sizeof(float) * max_points*128);
    float * output128_2;
    hipMalloc((void**)& output128_2, sizeof(float) * max_points*128);
    float * output1024;
    hipMalloc((void**)& output1024, sizeof(float) * max_points*1024);
    float * output1024_1;
    hipMalloc((void**)& output1024_1, sizeof(float) * max_points*1024);
    float * output1024_2;
    hipMalloc((void**)& output1024_2, sizeof(float) * max_points*1024);

	float *vector1024;
    hipMalloc((void**)& vector1024, sizeof(float) * 1024);
	float *vector512;
    hipMalloc((void**)& vector512, sizeof(float) * 512);
    float *vector512_1;
    hipMalloc((void**)& vector512_1, sizeof(float) * 512);
    float *vector512_2;
    hipMalloc((void**)& vector512_2, sizeof(float) * 512);
	float *vector256;
    hipMalloc((void**)& vector256, sizeof(float) * 256);
    float *vector256_1;
    hipMalloc((void**)& vector256_1, sizeof(float) * 256);
    float *vector256_2;
    hipMalloc((void**)& vector256_2, sizeof(float) * 256);
	float *vector9;
    hipMalloc((void**)& vector9, sizeof(float) * 9);
	float *vector64X64;
    hipMalloc((void**)& vector64X64, sizeof(float) * 64 * 64);
    float *vector10;
    hipMalloc((void**)& vector10, sizeof(float) * 10);

    //******************************************************************STN3d************************************************************************//
    //******************************************************************matrixTranspose*******************************************************************//

    
    //******************************************************************feat_stn_conv1*******************************************************************//
    int feat_stn_conv1_in_channels = 3;     // 输入通道数量
    int feat_stn_conv1_out_channels = 64;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_stn_conv1_weight;
    float* feat_stn_conv1_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_stn_conv1_weight, feat_stn_conv1_in_channels * feat_stn_conv1_out_channels * sizeof(float)); // 3 x 64 = 192
    hipMalloc((void**)&feat_stn_conv1_bias, feat_stn_conv1_out_channels * sizeof(float));                 // 64

    // 将这些数据从 host 拷贝到 device

    wbCheck(hipMemcpy(feat_stn_conv1_weight, &feat_stn_conv1_weight_host[0], feat_stn_conv1_in_channels * feat_stn_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_conv1_bias, &feat_stn_conv1_bias_host[0], feat_stn_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_stn_bn1*******************************************************************//
    int feat_stn_bn1_channels = 64;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_bn1_weight, *feat_stn_bn1_bias, *feat_stn_bn1_mean, *feat_stn_bn1_var;
    hipMalloc((void**)&feat_stn_bn1_weight, feat_stn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn1_bias, feat_stn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn1_mean, feat_stn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn1_var, feat_stn_bn1_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_bn1_weight, &feat_stn_bn1_weight_host[0], feat_stn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn1_bias, &feat_stn_bn1_bias_host[0], feat_stn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn1_mean, &feat_stn_bn1_mean_host[0], feat_stn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn1_var, &feat_stn_bn1_var_host[0], feat_stn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));




    // //******************************************************************feat_stn_relu1*******************************************************************//
    int feat_stn_relu1_channels = 64;  // 输出通道数量


    //******************************************************************feat_stn_conv2*******************************************************************//
    int feat_stn_conv2_in_channels = 64;     // 输入通道数量
    int feat_stn_conv2_out_channels = 128;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_stn_conv2_weight;
    float* feat_stn_conv2_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_stn_conv2_weight, feat_stn_conv2_in_channels * feat_stn_conv2_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_stn_conv2_bias, feat_stn_conv2_out_channels * sizeof(float));                 

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_stn_conv2_weight, &feat_stn_conv2_weight_host[0], feat_stn_conv2_in_channels * feat_stn_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_conv2_bias, &feat_stn_conv2_bias_host[0], feat_stn_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_stn_bn2*******************************************************************//
    int feat_stn_bn2_channels = 128;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_bn2_weight, *feat_stn_bn2_bias, *feat_stn_bn2_mean, *feat_stn_bn2_var;
    hipMalloc((void**)&feat_stn_bn2_weight, feat_stn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn2_bias, feat_stn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn2_mean, feat_stn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn2_var, feat_stn_bn2_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_bn2_weight, &feat_stn_bn2_weight_host[0], feat_stn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn2_bias, &feat_stn_bn2_bias_host[0], feat_stn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn2_mean, &feat_stn_bn2_mean_host[0], feat_stn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn2_var, &feat_stn_bn2_var_host[0], feat_stn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_stn_relu2*******************************************************************//
    int feat_stn_relu2_channels = 128;  // 输出通道数量

//******************************************************************feat_stn_conv3*******************************************************************//
    int feat_stn_conv3_in_channels = 128;     // 输入通道数量
    int feat_stn_conv3_out_channels = 1024;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_stn_conv3_weight;
    float* feat_stn_conv3_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_stn_conv3_weight, feat_stn_conv3_in_channels * feat_stn_conv3_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_stn_conv3_bias, feat_stn_conv3_out_channels * sizeof(float));                 

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_stn_conv3_weight, &feat_stn_conv3_weight_host[0], feat_stn_conv3_in_channels * feat_stn_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_conv3_bias, &feat_stn_conv3_bias_host[0], feat_stn_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************feat_stn_bn3*******************************************************************//
    int feat_stn_bn3_channels = 1024;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_bn3_weight, *feat_stn_bn3_bias, *feat_stn_bn3_mean, *feat_stn_bn3_var;
    hipMalloc((void**)&feat_stn_bn3_weight, feat_stn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn3_bias, feat_stn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn3_mean, feat_stn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn3_var, feat_stn_bn3_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_bn3_weight, &feat_stn_bn3_weight_host[0], feat_stn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn3_bias, &feat_stn_bn3_bias_host[0], feat_stn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn3_mean, &feat_stn_bn3_mean_host[0], feat_stn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn3_var, &feat_stn_bn3_var_host[0], feat_stn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_stn_relu3*******************************************************************//
    int feat_stn_relu3_channels = 1024;  // 输出通道数量


    // //******************************************************************feat_stn_max*******************************************************************//
    int feat_stn_max_channels = 1024;  // 输出通道数量



    // //******************************************************************feat_stn_fc1*******************************************************************//
    int feat_stn_fc1_in_channels = 1024;  // 输入通道数量
    int feat_stn_fc1_out_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_fc1_weight;
    float *feat_stn_fc1_bias;
    hipMalloc((void**)&feat_stn_fc1_weight, feat_stn_fc1_out_channels * feat_stn_fc1_in_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_fc1_bias, feat_stn_fc1_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_fc1_weight, &feat_stn_fc1_weight_host[0], feat_stn_fc1_out_channels * feat_stn_fc1_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_fc1_bias, &feat_stn_fc1_bias_host[0], feat_stn_fc1_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_stn_bn4*******************************************************************//
    int feat_stn_bn4_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_bn4_weight, *feat_stn_bn4_bias, *feat_stn_bn4_mean, *feat_stn_bn4_var;
    hipMalloc((void**)&feat_stn_bn4_weight, feat_stn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn4_bias, feat_stn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn4_mean, feat_stn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn4_var, feat_stn_bn4_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_bn4_weight, &feat_stn_bn4_weight_host[0], feat_stn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn4_bias, &feat_stn_bn4_bias_host[0], feat_stn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn4_mean, &feat_stn_bn4_mean_host[0], feat_stn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn4_var, &feat_stn_bn4_var_host[0], feat_stn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_stn_relu4*******************************************************************//
    int feat_stn_relu4_channels = 512;  // 输出通道数量




    //******************************************************************feat_stn_fc2*******************************************************************//
    int feat_stn_fc2_in_channels = 512;  // 输入通道数量
    int feat_stn_fc2_out_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_fc2_weight;
    float *feat_stn_fc2_bias;
    hipMalloc((void**)&feat_stn_fc2_weight, feat_stn_fc2_out_channels * feat_stn_fc2_in_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_fc2_bias, feat_stn_fc2_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_fc2_weight, &feat_stn_fc2_weight_host[0], feat_stn_fc2_out_channels * feat_stn_fc2_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_fc2_bias, &feat_stn_fc2_bias_host[0], feat_stn_fc2_out_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_stn_bn5*******************************************************************//
    int feat_stn_bn5_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_bn5_weight, *feat_stn_bn5_bias, *feat_stn_bn5_mean, *feat_stn_bn5_var;
    hipMalloc((void**)&feat_stn_bn5_weight, feat_stn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn5_bias, feat_stn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn5_mean, feat_stn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_bn5_var, feat_stn_bn5_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_bn5_weight, &feat_stn_bn5_weight_host[0], feat_stn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn5_bias, &feat_stn_bn5_bias_host[0], feat_stn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn5_mean, &feat_stn_bn5_mean_host[0], feat_stn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_bn5_var, &feat_stn_bn5_var_host[0], feat_stn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_stn_relu5*******************************************************************//
    int feat_stn_relu5_channels = 256;  // 输出通道数量



    //******************************************************************feat_stn_fc3*******************************************************************//
    int feat_stn_fc3_in_channels = 256;  // 输入通道数量
    int feat_stn_fc3_out_channels = 9;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_stn_fc3_weight;
    float *feat_stn_fc3_bias;
    hipMalloc((void**)&feat_stn_fc3_weight, feat_stn_fc3_out_channels * feat_stn_fc3_in_channels * sizeof(float));
    hipMalloc((void**)&feat_stn_fc3_bias, feat_stn_fc3_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_stn_fc3_weight, &feat_stn_fc3_weight_host[0], feat_stn_fc3_out_channels * feat_stn_fc3_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_stn_fc3_bias, &feat_stn_fc3_bias_host[0], feat_stn_fc3_out_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_stn_iden*******************************************************************//
    int feat_stn_iden_h = 3;  // 输出矩阵高度
    int feat_stn_iden_w = 3;  // 输出矩阵宽度


    
    //******************************************************************end STNkd*******************************************************************//


    // //******************************************************************matrixMultiply1*******************************************************************//




    //******************************************************************matrixTranspose2*******************************************************************//



//******************************************************************feat_conv1*******************************************************************//
    int feat_conv1_in_channels = 3;     // 输入通道数量
    int feat_conv1_out_channels = 64;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_conv1_weight;
    float* feat_conv1_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_conv1_weight, feat_conv1_in_channels * feat_conv1_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_conv1_bias, feat_conv1_out_channels * sizeof(float));                

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_conv1_weight, &feat_conv1_weight_host[0], feat_conv1_in_channels * feat_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_conv1_bias, &feat_conv1_bias_host[0], feat_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_bn1*******************************************************************//
    int feat_bn1_channels = 64;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_bn1_weight, *feat_bn1_bias, *feat_bn1_mean, *feat_bn1_var;
    hipMalloc((void**)&feat_bn1_weight, feat_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_bn1_bias, feat_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_bn1_mean, feat_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_bn1_var, feat_bn1_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_bn1_weight, &feat_bn1_weight_host[0], feat_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn1_bias, &feat_bn1_bias_host[0], feat_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn1_mean, &feat_bn1_mean_host[0], feat_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn1_var, &feat_bn1_var_host[0], feat_bn1_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_relu1*******************************************************************//
    int feat_relu1_channels = 64;  // 输出通道数量


    //******************************************************************STNkd************************************************************************//

//******************************************************************feat_fstn_conv1*******************************************************************//
    int feat_fstn_conv1_in_channels = 64;     // 输入通道数量
    int feat_fstn_conv1_out_channels = 64;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_fstn_conv1_weight;
    float* feat_fstn_conv1_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_fstn_conv1_weight, feat_fstn_conv1_in_channels * feat_fstn_conv1_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_fstn_conv1_bias, feat_fstn_conv1_out_channels * sizeof(float));                

    // 将这些数据从 host 拷贝到 device

    wbCheck(hipMemcpy(feat_fstn_conv1_weight, &feat_fstn_conv1_weight_host[0], feat_fstn_conv1_in_channels * feat_fstn_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_conv1_bias, &feat_fstn_conv1_bias_host[0], feat_fstn_conv1_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_bn1*******************************************************************//
    int feat_fstn_bn1_channels = 64;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_bn1_weight, *feat_fstn_bn1_bias, *feat_fstn_bn1_mean, *feat_fstn_bn1_var;
    hipMalloc((void**)&feat_fstn_bn1_weight, feat_fstn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn1_bias, feat_fstn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn1_mean, feat_fstn_bn1_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn1_var, feat_fstn_bn1_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_bn1_weight, &feat_fstn_bn1_weight_host[0], feat_fstn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn1_bias, &feat_fstn_bn1_bias_host[0], feat_fstn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn1_mean, &feat_fstn_bn1_mean_host[0], feat_fstn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn1_var, &feat_fstn_bn1_var_host[0], feat_fstn_bn1_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_relu1*******************************************************************//
    int feat_fstn_relu1_channels = 64;  // 输出通道数量


    //******************************************************************feat_fstn_conv2*******************************************************************//
    int feat_fstn_conv2_in_channels = 64;     // 输入通道数量
    int feat_fstn_conv2_out_channels = 128;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_fstn_conv2_weight;
    float* feat_fstn_conv2_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_fstn_conv2_weight, feat_fstn_conv2_in_channels * feat_fstn_conv2_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_fstn_conv2_bias, feat_fstn_conv2_out_channels * sizeof(float));                 

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_fstn_conv2_weight, &feat_fstn_conv2_weight_host[0], feat_fstn_conv2_in_channels * feat_fstn_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_conv2_bias, &feat_fstn_conv2_bias_host[0], feat_fstn_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_bn2*******************************************************************//
    int feat_fstn_bn2_channels = 128;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_bn2_weight, *feat_fstn_bn2_bias, *feat_fstn_bn2_mean, *feat_fstn_bn2_var;
    hipMalloc((void**)&feat_fstn_bn2_weight, feat_fstn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn2_bias, feat_fstn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn2_mean, feat_fstn_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn2_var, feat_fstn_bn2_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_bn2_weight, &feat_fstn_bn2_weight_host[0], feat_fstn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn2_bias, &feat_fstn_bn2_bias_host[0], feat_fstn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn2_mean, &feat_fstn_bn2_mean_host[0], feat_fstn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn2_var, &feat_fstn_bn2_var_host[0], feat_fstn_bn2_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_relu2*******************************************************************//
    int feat_fstn_relu2_channels = 128;  // 输出通道数量


//******************************************************************feat_fstn_conv3*******************************************************************//
    int feat_fstn_conv3_in_channels = 128;     // 输入通道数量
    int feat_fstn_conv3_out_channels = 1024;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_fstn_conv3_weight;
    float* feat_fstn_conv3_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_fstn_conv3_weight, feat_fstn_conv3_in_channels * feat_fstn_conv3_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_fstn_conv3_bias, feat_fstn_conv3_out_channels * sizeof(float));                 

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_fstn_conv3_weight, &feat_fstn_conv3_weight_host[0], feat_fstn_conv3_in_channels * feat_fstn_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_conv3_bias, &feat_fstn_conv3_bias_host[0], feat_fstn_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************feat_fstn_bn3*******************************************************************//
    int feat_fstn_bn3_channels = 1024;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_bn3_weight, *feat_fstn_bn3_bias, *feat_fstn_bn3_mean, *feat_fstn_bn3_var;
    hipMalloc((void**)&feat_fstn_bn3_weight, feat_fstn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn3_bias, feat_fstn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn3_mean, feat_fstn_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn3_var, feat_fstn_bn3_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_bn3_weight, &feat_fstn_bn3_weight_host[0], feat_fstn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn3_bias, &feat_fstn_bn3_bias_host[0], feat_fstn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn3_mean, &feat_fstn_bn3_mean_host[0], feat_fstn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn3_var, &feat_fstn_bn3_var_host[0], feat_fstn_bn3_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************feat_fstn_relu3*******************************************************************//
    int feat_fstn_relu3_channels = 1024;  // 输出通道数量

    // //******************************************************************feat_fstn_max*******************************************************************//
    int feat_fstn_max_channels = 1024;  // 输出通道数量



    // //******************************************************************feat_fstn_fc1*******************************************************************//
    int feat_fstn_fc1_in_channels = 1024;  // 输入通道数量
    int feat_fstn_fc1_out_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_fc1_weight;
    float *feat_fstn_fc1_bias;
    hipMalloc((void**)&feat_fstn_fc1_weight, feat_fstn_fc1_out_channels * feat_fstn_fc1_in_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_fc1_bias, feat_fstn_fc1_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_fc1_weight, &feat_fstn_fc1_weight_host[0], feat_fstn_fc1_out_channels * feat_fstn_fc1_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_fc1_bias, &feat_fstn_fc1_bias_host[0], feat_fstn_fc1_out_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_fstn_bn4*******************************************************************//
    int feat_fstn_bn4_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_bn4_weight, *feat_fstn_bn4_bias, *feat_fstn_bn4_mean, *feat_fstn_bn4_var;
    hipMalloc((void**)&feat_fstn_bn4_weight, feat_fstn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn4_bias, feat_fstn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn4_mean, feat_fstn_bn4_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn4_var, feat_fstn_bn4_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_bn4_weight, &feat_fstn_bn4_weight_host[0], feat_fstn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn4_bias, &feat_fstn_bn4_bias_host[0], feat_fstn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn4_mean, &feat_fstn_bn4_mean_host[0], feat_fstn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn4_var, &feat_fstn_bn4_var_host[0], feat_fstn_bn4_channels * sizeof(float), hipMemcpyHostToDevice));



    // //******************************************************************feat_fstn_relu4*******************************************************************//
    int feat_fstn_relu4_channels = 512;  // 输出通道数量


    //******************************************************************feat_fstn_fc2*******************************************************************//
    int feat_fstn_fc2_in_channels = 512;  // 输入通道数量
    int feat_fstn_fc2_out_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_fc2_weight;
    float *feat_fstn_fc2_bias;
    hipMalloc((void**)&feat_fstn_fc2_weight, feat_fstn_fc2_out_channels * feat_fstn_fc2_in_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_fc2_bias, feat_fstn_fc2_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_fc2_weight, &feat_fstn_fc2_weight_host[0], feat_fstn_fc2_out_channels * feat_fstn_fc2_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_fc2_bias, &feat_fstn_fc2_bias_host[0], feat_fstn_fc2_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_bn5*******************************************************************//
    int feat_fstn_bn5_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_bn5_weight, *feat_fstn_bn5_bias, *feat_fstn_bn5_mean, *feat_fstn_bn5_var;
    hipMalloc((void**)&feat_fstn_bn5_weight, feat_fstn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn5_bias, feat_fstn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn5_mean, feat_fstn_bn5_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_bn5_var, feat_fstn_bn5_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_bn5_weight, &feat_fstn_bn5_weight_host[0], feat_fstn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn5_bias, &feat_fstn_bn5_bias_host[0], feat_fstn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn5_mean, &feat_fstn_bn5_mean_host[0], feat_fstn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_bn5_var, &feat_fstn_bn5_var_host[0], feat_fstn_bn5_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_fstn_relu5*******************************************************************//
    int feat_fstn_relu5_channels = 256;  // 输出通道数量



    //******************************************************************feat_fstn_fc3*******************************************************************//
    int feat_fstn_fc3_in_channels = 256;  // 输入通道数量
    int feat_fstn_fc3_out_channels = 64*64;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_fstn_fc3_weight;
    float *feat_fstn_fc3_bias;
    hipMalloc((void**)&feat_fstn_fc3_weight, feat_fstn_fc3_out_channels * feat_fstn_fc3_in_channels * sizeof(float));
    hipMalloc((void**)&feat_fstn_fc3_bias, feat_fstn_fc3_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_fstn_fc3_weight, &feat_fstn_fc3_weight_host[0], feat_fstn_fc3_out_channels * feat_fstn_fc3_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_fstn_fc3_bias, &feat_fstn_fc3_bias_host[0], feat_fstn_fc3_out_channels * sizeof(float), hipMemcpyHostToDevice));

    //******************************************************************feat_fstn_iden*******************************************************************//
    int feat_fstn_iden_h = 64;  // 输出矩阵高度
    int feat_fstn_iden_w = 64;  // 输出矩阵宽度



    //******************************************************************end STNkd*******************************************************************//
    
    //******************************************************************matrixTranspose3*******************************************************************//


    //******************************************************************matrixMultiply2*******************************************************************//



    //******************************************************************matrixTranspose4*******************************************************************//



    //******************************************************************feat_conv2*******************************************************************//
    int feat_conv2_in_channels = 64;     // 输入通道数量
    int feat_conv2_out_channels = 128;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_conv2_weight;
    float* feat_conv2_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_conv2_weight, feat_conv2_in_channels * feat_conv2_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_conv2_bias, feat_conv2_out_channels * sizeof(float));                

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_conv2_weight, &feat_conv2_weight_host[0], feat_conv2_in_channels * feat_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_conv2_bias, &feat_conv2_bias_host[0], feat_conv2_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_bn2*******************************************************************//
    int feat_bn2_channels = 128;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_bn2_weight, *feat_bn2_bias, *feat_bn2_mean, *feat_bn2_var;
    hipMalloc((void**)&feat_bn2_weight, feat_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_bn2_bias, feat_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_bn2_mean, feat_bn2_channels * sizeof(float));
    hipMalloc((void**)&feat_bn2_var, feat_bn2_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_bn2_weight, &feat_bn2_weight_host[0], feat_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn2_bias, &feat_bn2_bias_host[0], feat_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn2_mean, &feat_bn2_mean_host[0], feat_bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn2_var, &feat_bn2_var_host[0], feat_bn2_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************feat_relu2*******************************************************************//
    int feat_relu2_channels = 128;  // 输出通道数量

//******************************************************************feat_conv3*******************************************************************//
    int feat_conv3_in_channels = 128;     // 输入通道数量
    int feat_conv3_out_channels = 1024;   // 输出通道数量

    // 为输入、输出、权重和偏置分配内存
    float* feat_conv3_weight;
    float* feat_conv3_bias;

    // CUDA 设备内存分配
    hipMalloc((void**)&feat_conv3_weight, feat_conv3_in_channels * feat_conv3_out_channels * sizeof(float)); 
    hipMalloc((void**)&feat_conv3_bias, feat_conv3_out_channels * sizeof(float));                

    // 将这些数据从 host 拷贝到 device
    wbCheck(hipMemcpy(feat_conv3_weight, &feat_conv3_weight_host[0], feat_conv3_in_channels * feat_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_conv3_bias, &feat_conv3_bias_host[0], feat_conv3_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************feat_bn3*******************************************************************//
    int feat_bn3_channels = 1024;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *feat_bn3_weight, *feat_bn3_bias, *feat_bn3_mean, *feat_bn3_var;
    hipMalloc((void**)&feat_bn3_weight, feat_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_bn3_bias, feat_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_bn3_mean, feat_bn3_channels * sizeof(float));
    hipMalloc((void**)&feat_bn3_var, feat_bn3_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(feat_bn3_weight, &feat_bn3_weight_host[0], feat_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn3_bias, &feat_bn3_bias_host[0], feat_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn3_mean, &feat_bn3_mean_host[0], feat_bn3_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(feat_bn3_var, &feat_bn3_var_host[0], feat_bn3_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************feat_max*******************************************************************//
    int feat_max_channels = 1024;  // 输出通道数量


    //******************************************************************fc1*******************************************************************//
    int fc1_in_channels = 1024;  // 输入通道数量
    int fc1_out_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *fc1_weight;
    float *fc1_bias;
    hipMalloc((void**)&fc1_weight, fc1_out_channels * fc1_in_channels * sizeof(float));
    hipMalloc((void**)&fc1_bias, fc1_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(fc1_weight, &fc1_weight_host_host[0], fc1_out_channels * fc1_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(fc1_bias, &fc1_bias_host_host[0], fc1_out_channels * sizeof(float), hipMemcpyHostToDevice));

    // //******************************************************************bn1*******************************************************************//
    int bn1_channels = 512;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *bn1_weight, *bn1_bias, *bn1_mean, *bn1_var;
    hipMalloc((void**)&bn1_weight, bn1_channels * sizeof(float));
    hipMalloc((void**)&bn1_bias, bn1_channels * sizeof(float));
    hipMalloc((void**)&bn1_mean, bn1_channels * sizeof(float));
    hipMalloc((void**)&bn1_var, bn1_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(bn1_weight, &bn1_weight_host[0], bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn1_bias, &bn1_bias_host[0], bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn1_mean, &bn1_mean_host[0], bn1_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn1_var, &bn1_var_host[0], bn1_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************relu1*******************************************************************//
    int relu1_channels = 512;  // 输出通道数量


    //******************************************************************fc2*******************************************************************//
    int fc2_in_channels = 512;  // 输入通道数量
    int fc2_out_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *fc2_weight;
    float *fc2_bias;
    hipMalloc((void**)&fc2_weight, fc2_out_channels * fc2_in_channels * sizeof(float));
    hipMalloc((void**)&fc2_bias, fc2_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(fc2_weight, &fc2_weight_host[0], fc2_out_channels * fc2_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(fc2_bias, &fc2_bias_host[0], fc2_out_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************bn2*******************************************************************//
    int bn2_channels = 256;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *bn2_weight, *bn2_bias, *bn2_mean, *bn2_var;
    hipMalloc((void**)&bn2_weight, bn2_channels * sizeof(float));
    hipMalloc((void**)&bn2_bias, bn2_channels * sizeof(float));
    hipMalloc((void**)&bn2_mean, bn2_channels * sizeof(float));
    hipMalloc((void**)&bn2_var, bn2_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(bn2_weight, &bn2_weight_host[0], bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn2_bias, &bn2_bias_host[0], bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn2_mean, &bn2_mean_host[0], bn2_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(bn2_var, &bn2_var_host[0], bn2_channels * sizeof(float), hipMemcpyHostToDevice));


    // //******************************************************************relu2*******************************************************************//
    int relu2_channels = 256;  // 输出通道数量



    //******************************************************************fc3*******************************************************************//
    int fc3_in_channels = 256;  // 输入通道数量
    int fc3_out_channels = 10;  // 输出通道数量

    // 分配 CUDA 设备内存
    float *fc3_weight;
    float *fc3_bias;
    hipMalloc((void**)&fc3_weight, fc3_out_channels * fc3_in_channels * sizeof(float));
    hipMalloc((void**)&fc3_bias, fc3_out_channels * sizeof(float));

    // 从主机拷贝数据到设备
    wbCheck(hipMemcpy(fc3_weight, &fc3_weight_host[0], fc3_out_channels * fc3_in_channels * sizeof(float), hipMemcpyHostToDevice));
    wbCheck(hipMemcpy(fc3_bias, &fc3_bias_host[0], fc3_out_channels * sizeof(float), hipMemcpyHostToDevice));




    // 开始计时，使用chrono计时，不支持其它计时方式
    auto start = std::chrono::high_resolution_clock::now();
    
    for (size_t i = 0; i < list_of_points.size(); i++)
    {
        // TODO ...在这里实现利用CUDA对点云数据进行深度学习的推理过程，当然，你也可以改进for循环以使用batch推理提速...
        // 打印每一帧的数据，仅用于调试！
    
        // std::cout << "Points " << i << ": ";
        // // for (const auto& point : list_of_points[i]) {
        // //     std::cout << point << " ";
        // // }
        // std::cout << "\nLabel: " << list_of_labels[i] << std::endl;



        int num_points=list_of_points[i].size()/3;
        
        const int Grid=512;
        const int Block3=(num_points *3 +Grid -1 )/Grid; //3*num_points
        const int Block64=(num_points *64 +Grid -1 )/Grid;
        const int Block128=(num_points *128 +Grid -1 )/Grid;
        const int Block1024=(num_points *1024 +Grid -1 )/Grid;

        size_t sharedMemSize = (num_points + 1024) * sizeof(float);

        // 调用 CUDA核函数
        //matrixTranspose1
        wbCheck(hipMemcpy(input, list_of_points[i].data(), num_points * 3 * sizeof(float), hipMemcpyHostToDevice));

        matrixTranspose<<<Block3, Grid>>>(input, Tinput,num_points,3);
        //wbCheck(hipGetLastError());
        //wbCheck(hipDeviceSynchronize());
    

        //******************************************************************STN3d************************************************************************//
        //feat_stn_conv1
        Convolution<<<Block64, Grid, sharedMemSize>>>(Tinput, output64, feat_stn_conv1_weight, feat_stn_conv1_bias, num_points, feat_stn_conv1_in_channels, feat_stn_conv1_out_channels);

        //feat_stn_bn1
        BatchNorm<<<Block64, Grid>>>(output64, output64_2, feat_stn_bn1_weight,  feat_stn_bn1_bias, feat_stn_bn1_mean, feat_stn_bn1_var,num_points, feat_stn_bn1_channels);

        //feat_stn_relu1
        Relu<<<Block64, Grid>>>(output64_2, output64_3, num_points, feat_stn_relu1_channels);

        //feat_stn_conv2
        Convolution<<<Block128, Grid, sharedMemSize>>>(output64_3, output128, feat_stn_conv2_weight, feat_stn_conv2_bias, num_points, feat_stn_conv2_in_channels, feat_stn_conv2_out_channels);
    
        //feat_stn_bn2
        BatchNorm<<<Block128, Grid>>>(output128, output128_1, feat_stn_bn2_weight,  feat_stn_bn2_bias, feat_stn_bn2_mean, feat_stn_bn2_var,num_points, feat_stn_bn2_channels);

        //feat_stn_relu2
        Relu<<<Block128, Grid>>>(output128_1, output128_2, num_points, feat_stn_relu2_channels);
    
        //feat_stn_conv3
        Convolution<<<Block1024, Grid, sharedMemSize>>>(output128_2, output1024, feat_stn_conv3_weight, feat_stn_conv3_bias, num_points, feat_stn_conv3_in_channels, feat_stn_conv3_out_channels);
    
        //feat_stn_bn3
        BatchNorm<<<Block1024, Grid>>>(output1024, output1024_1, feat_stn_bn3_weight,  feat_stn_bn3_bias, feat_stn_bn3_mean, feat_stn_bn3_var,num_points, feat_stn_bn3_channels);

        //feat_stn_relu3
        Relu<<<Block1024, Grid>>>(output1024_1, output1024_2, num_points, feat_stn_relu3_channels);
  
        //feat_stn_max
        MaxPool<<<1, 1024>>>(output1024_2, vector1024, num_points, feat_stn_max_channels);
                
        // float* host_x;
        // host_x = (float*)malloc(1024* num_points*sizeof(float));
        // hipMemcpy(host_x, output1024_2,  1024* num_points*sizeof(float), hipMemcpyDeviceToHost);
        // for (int x = 0; x < num_points; x++){
        //     std::cout << host_x[x] << ",";

        // }
        //feat_stn_fc1
        FC<<<1, 512>>>(vector1024, vector512, feat_stn_fc1_weight, feat_stn_fc1_bias,feat_stn_fc1_in_channels,feat_stn_fc1_out_channels);

        //feat_stn_bn4
        BatchNorm<<<1, 512>>>(vector512, vector512_1, feat_stn_bn4_weight,  feat_stn_bn4_bias, feat_stn_bn4_mean, feat_stn_bn4_var,1, feat_stn_bn4_channels);

        //feat_stn_relu4
        Relu<<<1, 512>>>(vector512_1, vector512_2, 1, feat_stn_relu4_channels);

        //feat_stn_fc2
        FC<<<1, 256>>>(vector512_2, vector256, feat_stn_fc2_weight, feat_stn_fc2_bias,feat_stn_fc2_in_channels,feat_stn_fc2_out_channels);

        //feat_stn_bn5
        BatchNorm<<<1, 256>>>(vector256, vector256_1, feat_stn_bn5_weight,  feat_stn_bn5_bias, feat_stn_bn5_mean, feat_stn_bn5_var,1, feat_stn_bn5_channels);

        //feat_stn_relu5
        Relu<<<1, 256>>>(vector256_1, vector256_2, 1, feat_stn_relu5_channels);

        //feat_stn_fc3
        FC<<<1, 9>>>(vector256_2, vector9, feat_stn_fc3_weight, feat_stn_fc3_bias,feat_stn_fc3_in_channels,feat_stn_fc3_out_channels);

        //feat_stn_iden
        Iden<<<1, 9>>>(vector9, feat_stn_iden_h, feat_stn_iden_w);

        //matrixMultiply1
        matrixMultiply<<<Block3, Grid>>>(input, vector9, Tinput, num_points, 3, 3);

        //matrixTranspose2
 
        matrixTranspose<<<Block3, Grid>>>(Tinput, input,num_points,3);

        //feat_conv1
        Convolution<<<Block64, Grid, sharedMemSize>>>(input, Toutput64, feat_conv1_weight, feat_conv1_bias, num_points, feat_conv1_in_channels, feat_conv1_out_channels);
        
        //feat_bn1
        BatchNorm<<<Block64, Grid>>>(Toutput64, Toutput64_2, feat_bn1_weight,  feat_bn1_bias, feat_bn1_mean, feat_bn1_var,num_points, feat_bn1_channels);
    
        //feat_relu1
        Relu<<<Block64, Grid>>>(Toutput64_2, Toutput64_3, num_points, feat_relu1_channels);

        hipMemcpy(output64,Toutput64_3,num_points*64*sizeof(float),hipMemcpyDeviceToDevice);

        //******************************************************************STNkd************************************************************************//
        //feat_fstn_conv1
        Convolution<<<Block64, Grid, sharedMemSize>>>(Toutput64_3, Toutput64_2, feat_fstn_conv1_weight, feat_fstn_conv1_bias, num_points, feat_fstn_conv1_in_channels, feat_fstn_conv1_out_channels);
        
        //feat_fstn_bn1
        BatchNorm<<<Block64, Grid>>>(Toutput64_2, Toutput64, feat_fstn_bn1_weight,  feat_fstn_bn1_bias, feat_fstn_bn1_mean, feat_fstn_bn1_var,num_points, feat_fstn_bn1_channels);
    
        //feat_fstn_relu1
        Relu<<<Block64, Grid>>>(Toutput64, Toutput64_3, num_points, feat_fstn_relu1_channels);
    
        //feat_fstn_conv2
        Convolution<<<Block128, Grid, sharedMemSize>>>(Toutput64_3, output128, feat_fstn_conv2_weight, feat_fstn_conv2_bias, num_points, feat_fstn_conv2_in_channels, feat_fstn_conv2_out_channels);
    
        //feat_fstn_bn2
        BatchNorm<<<Block128, Grid>>>(output128, output128_1, feat_fstn_bn2_weight,  feat_fstn_bn2_bias, feat_fstn_bn2_mean, feat_fstn_bn2_var,num_points, feat_fstn_bn2_channels);

        //feat_fstn_relu2
        Relu<<<Block128, Grid>>>(output128_1, output128_2, num_points, feat_fstn_relu2_channels);
    
        //feat_fstn_conv3
        Convolution<<<Block1024, Grid, sharedMemSize>>>(output128_2, output1024, feat_fstn_conv3_weight, feat_fstn_conv3_bias, num_points, feat_fstn_conv3_in_channels, feat_fstn_conv3_out_channels);
    
        //feat_fstn_bn3
        BatchNorm<<<Block1024, Grid>>>(output1024, output1024_1, feat_fstn_bn3_weight,  feat_fstn_bn3_bias, feat_fstn_bn3_mean, feat_fstn_bn3_var,num_points, feat_fstn_bn3_channels);

        //feat_fstn_relu3
        Relu<<<Block1024, Grid>>>(output1024_1, output1024_2, num_points, feat_fstn_relu3_channels);

        //feat_fstn_max
        MaxPool<<<1, 1024>>>(output1024_2, vector1024, num_points, feat_fstn_max_channels);

        //feat_fstn_fc1
        FC<<<1, 512>>>(vector1024, vector512, feat_fstn_fc1_weight, feat_fstn_fc1_bias,feat_fstn_fc1_in_channels,feat_fstn_fc1_out_channels);

        //feat_fstn_bn4
        BatchNorm<<<1, 512>>>(vector512, vector512_1, feat_fstn_bn4_weight,  feat_fstn_bn4_bias, feat_fstn_bn4_mean, feat_fstn_bn4_var,1, feat_fstn_bn4_channels);

        //feat_fstn_relu4
        Relu<<<1, 512>>>(vector512_1, vector512_2, 1, feat_fstn_relu4_channels);

        //feat_fstn_fc2
        FC<<<1, 256>>>(vector512_2, vector256, feat_fstn_fc2_weight, feat_fstn_fc2_bias,feat_fstn_fc2_in_channels,feat_fstn_fc2_out_channels);

        //feat_fstn_bn5
        BatchNorm<<<1, 256>>>(vector256, vector256_1, feat_fstn_bn5_weight,  feat_fstn_bn5_bias, feat_fstn_bn5_mean, feat_fstn_bn5_var,1, feat_fstn_bn5_channels);

        //feat_fstn_relu5
        Relu<<<1, 256>>>(vector256_1, vector256_2, 1, feat_fstn_relu5_channels);

        //feat_fstn_fc3
        FC<<<4, 1024>>>(vector256_2, vector64X64, feat_fstn_fc3_weight, feat_fstn_fc3_bias,feat_fstn_fc3_in_channels,feat_fstn_fc3_out_channels);

        //feat_fstn_iden
        Iden<<<4, 1024>>>(vector64X64, feat_fstn_iden_h, feat_fstn_iden_w);

        //******************************************************************end STNkd************************************************************************//
        //matrixTranspose3
        matrixTranspose<<<Block64, Grid>>>(output64, Toutput64,64,num_points);
        
        //matrixMultiply2
        matrixMultiply<<<Block64, Grid>>>(Toutput64, vector64X64, output64, num_points, 64, 64);

        //matrixTranspose4
        matrixTranspose<<<Block64, Grid>>>(output64, Toutput64,num_points,64);

        //feat_conv2
        Convolution<<<Block128, Grid, sharedMemSize>>>(Toutput64, output128, feat_conv2_weight, feat_conv2_bias, num_points, feat_conv2_in_channels, feat_conv2_out_channels);
        
        //feat_bn2
        BatchNorm<<<Block128, Grid>>>(output128, output128_1, feat_bn2_weight,  feat_bn2_bias, feat_bn2_mean, feat_bn2_var,num_points, feat_bn2_channels);
    
        //feat_relu2
        Relu<<<Block128, Grid>>>(output128_1, output128_2, num_points, feat_relu2_channels);

        //feat_conv3
        Convolution<<<Block1024, Grid, sharedMemSize>>>(output128_2, output1024, feat_conv3_weight, feat_conv3_bias, num_points, feat_conv3_in_channels, feat_conv3_out_channels);
        
        //feat_bn3
        BatchNorm<<<Block1024, Grid>>>(output1024, output1024_1, feat_bn3_weight,  feat_bn3_bias, feat_bn3_mean, feat_bn3_var,num_points, feat_bn3_channels);
    
        //feat_max
        MaxPool<<<1, 1024>>>(output1024_1, vector1024, num_points, feat_max_channels);

        //******************************************************************end PointNetEncoder************************************************************************//

        //fc1
        FC<<<1, 512>>>(vector1024, vector512, fc1_weight, fc1_bias, fc1_in_channels, fc1_out_channels);

        //bn1
        BatchNorm<<<1, 512>>>(vector512, vector512_1, bn1_weight,  bn1_bias, bn1_mean, bn1_var,1, bn1_channels);

        //relu1
        Relu<<<1, 512>>>(vector512_1, vector512_2, 1, relu1_channels);


        //fc2
        FC<<<1, 256>>>(vector512_2, vector256, fc2_weight, fc2_bias, fc2_in_channels, fc2_out_channels);

        //bn2
        BatchNorm<<<1, 256>>>(vector256, vector256_1, bn2_weight,  bn2_bias, bn2_mean, bn2_var,1, bn2_channels);

        //relu2
        Relu<<<1, 256>>>(vector256_1, vector256_2, 1, relu2_channels);

        //fc3
        FC<<<1, 10>>>(vector256_2, vector10, fc3_weight, fc3_bias, fc3_in_channels, fc3_out_channels);





        //获取推理结果
        float max = 0;
        float* host_output;
        host_output = (float*)malloc(10 * sizeof(float));
        hipMemcpy(host_output, vector10, 10 * sizeof(float), hipMemcpyDeviceToHost);
    
    
        for (int j = 0; j < 10; j++){
            std::cout << host_output[j] << ",";
            if(host_output[j] > max)
            {
                max = host_output[j];
                result = j;
            }
        }

        std::cout << std::endl;


        if(result == list_of_labels[i])
        {
            num++;
        }


        
    }
    
    // 向主机端同步以等待所有异步调用的GPU kernel执行完毕，这句必须要有
    hipDeviceSynchronize();

    // 结束计时
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end - start;

    // 输出结果，请严格保持此输出格式，并把0.0001替换成实际的准确率，请不要输出除了此结果之外的任何内容！！！
    std::cout << std::fixed << std::setprecision(4) << diff.count() << ":"<< num*1.0/list_of_points.size();




    // 清理分配的内存
    

    hipFree(feat_stn_conv1_weight);
    hipFree(feat_stn_conv1_bias);
    hipFree(feat_stn_bn1_weight);
    hipFree(feat_stn_bn1_bias);
    hipFree(feat_stn_bn1_mean);
    hipFree(feat_stn_bn1_var);

    hipFree(feat_stn_conv2_weight);
    hipFree(feat_stn_conv2_bias);
    hipFree(feat_stn_bn2_weight);
    hipFree(feat_stn_bn2_bias);
    hipFree(feat_stn_bn2_mean);
    hipFree(feat_stn_bn2_var);

    hipFree(feat_stn_conv3_weight);
    hipFree(feat_stn_conv3_bias);
    hipFree(feat_stn_bn3_weight);
    hipFree(feat_stn_bn3_bias);
    hipFree(feat_stn_bn3_mean);
    hipFree(feat_stn_bn3_var);

 
    hipFree(feat_stn_fc1_weight);
    hipFree(feat_stn_fc1_bias);

    hipFree(feat_stn_bn4_weight);
    hipFree(feat_stn_bn4_bias);
    hipFree(feat_stn_bn4_mean);
    hipFree(feat_stn_bn4_var);

    hipFree(feat_stn_fc2_weight);
    hipFree(feat_stn_fc2_bias);

    hipFree(feat_stn_bn5_weight);
    hipFree(feat_stn_bn5_bias);
    hipFree(feat_stn_bn5_mean);
    hipFree(feat_stn_bn5_var);



    hipFree(feat_stn_fc3_weight);
    hipFree(feat_stn_fc3_bias);


    

    hipFree(feat_conv1_weight);
    hipFree(feat_conv1_bias);



    hipFree(feat_fstn_conv1_weight);
    hipFree(feat_fstn_conv1_bias);

    hipFree(feat_fstn_bn1_weight);
    hipFree(feat_fstn_bn1_bias);
    hipFree(feat_fstn_bn1_mean);
    hipFree(feat_fstn_bn1_var);


    hipFree(feat_fstn_conv2_weight);
    hipFree(feat_fstn_conv2_bias);

    hipFree(feat_fstn_bn2_weight);
    hipFree(feat_fstn_bn2_bias);
    hipFree(feat_fstn_bn2_mean);
    hipFree(feat_fstn_bn2_var);



    hipFree(feat_fstn_conv3_weight);
    hipFree(feat_fstn_conv3_bias);

    hipFree(feat_fstn_bn3_weight);
    hipFree(feat_fstn_bn3_bias);
    hipFree(feat_fstn_bn3_mean);
    hipFree(feat_fstn_bn3_var);




    hipFree(feat_fstn_fc1_weight);
    hipFree(feat_fstn_fc1_bias);

    hipFree(feat_fstn_bn4_weight);
    hipFree(feat_fstn_bn4_bias);
    hipFree(feat_fstn_bn4_mean);
    hipFree(feat_fstn_bn4_var);



    hipFree(feat_fstn_fc2_weight);
    hipFree(feat_fstn_fc2_bias);

    hipFree(feat_fstn_bn5_weight);
    hipFree(feat_fstn_bn5_bias);
    hipFree(feat_fstn_bn5_mean);
    hipFree(feat_fstn_bn5_var);


 
    hipFree(feat_fstn_fc3_weight);
    hipFree(feat_fstn_fc3_bias);




    hipFree(feat_conv2_weight);
    hipFree(feat_conv2_bias);

    hipFree(feat_bn2_weight);
    hipFree(feat_bn2_bias);
    hipFree(feat_bn2_mean);
    hipFree(feat_bn2_var);


    hipFree(feat_conv3_weight);
    hipFree(feat_conv3_bias);

    hipFree(feat_bn3_weight);
    hipFree(feat_bn3_bias);
    hipFree(feat_bn3_mean);
    hipFree(feat_bn3_var);


    hipFree(fc1_weight);
    hipFree(fc1_bias);

    hipFree(bn1_weight);
    hipFree(bn1_bias);
    hipFree(bn1_mean);
    hipFree(bn1_var);

    hipFree(fc2_weight);
    hipFree(fc2_bias);

    hipFree(bn2_weight);
    hipFree(bn2_bias);
    hipFree(bn2_mean);
    hipFree(bn2_var);


    hipFree(fc3_weight);
    hipFree(fc3_bias);

	hipFree(input);
	hipFree(Tinput);
    hipFree(output64);
    hipFree(output64_2);
    hipFree(output64_3);   
    hipFree(Toutput64);    
    hipFree(Toutput64_2);
    hipFree(Toutput64_3);  
    hipFree(output128);
    hipFree(output128_1);
    hipFree(output128_2);
    hipFree(output1024);
    hipFree(output1024_1);
    hipFree(output1024_2);
    hipFree(vector1024);
    hipFree(vector512);
    hipFree(vector512_1);
    hipFree(vector512_2);
    hipFree(vector256);
    hipFree(vector256_1);
    hipFree(vector256_2);
    hipFree(vector9);
    hipFree(vector64X64);







    return 0;
}